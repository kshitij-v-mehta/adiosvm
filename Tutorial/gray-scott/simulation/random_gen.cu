#include "random_gen.h"

int curand_init_gen(double * devdata, int n, hiprandGenerator_t *gen) {
    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(gen, HIPRAND_RNG_PSEUDO_MT19937));
    
    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(*gen, 1234ULL));
}

int gen_curand_values(double *devdata, int n, hiprandGenerator_t gen) {
    /* Generate n doubles on device */
    CURAND_CALL(hiprandGenerateUniformDouble(gen, devdata, n));
    
    printf("Done with hiprand\n");
    fflush(stdout);
    return EXIT_SUCCESS;
}

int curand_cleanup(hiprandGenerator_t gen) {
    /* Cleanup */
    CURAND_CALL(hiprandDestroyGenerator(gen));
}

